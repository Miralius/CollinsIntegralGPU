﻿#include <iostream>
#include <vector>
#include <complex>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"


// Функция освобождения памяти GPU (device_pointer_input — указатель на входное поле, device_pointer_output — на выходное поле (результат), device_pointer_x1 — на вектор x, device_pointer_x2 — на вектор y, device_pointer_x3 — на вектор u или z, device_pointer_x4 — на вектор v, device_pointer_parameters — на вектор параметров преобразования, device_pointer_dimension — на вектор размерностей, device_pointer_progress — на атомарную переменную прогресса процесса)
void freeGPUMemory(hipDoubleComplex* device_pointer_input, hipDoubleComplex* device_pointer_output, double* device_pointer_x1, double* device_pointer_x2, double* device_pointer_x3, double* device_pointer_x4, double* device_pointer_parameters, int* device_pointer_dimension, int* device_pointer_progress) {
    hipFree(device_pointer_input);
    hipFree(device_pointer_output);
    hipFree(device_pointer_x1);
    hipFree(device_pointer_x2);
    hipFree(device_pointer_x3);
    hipFree(device_pointer_x4);
    hipFree(device_pointer_parameters);
    hipFree(device_pointer_dimension);
    hipFree(device_pointer_progress);
}

// Сложение двух CUDA double комплексных чисел (left — число слева, right — справа)
__device__ hipDoubleComplex operator+(const hipDoubleComplex& left, const hipDoubleComplex& right) {
    return hipCadd(left, right);
}

// Сложение двух CUDA double комплексных чисел с присвоением (left — переменная до знака "=", right — после)
__device__ hipDoubleComplex operator+=(hipDoubleComplex& left, const hipDoubleComplex& right) {
    left = left + right;
    return left;
}

// Перемножение двух CUDA double комплексных чисел (left — число слева, right — справа)
__device__ hipDoubleComplex operator*(const hipDoubleComplex& left, const hipDoubleComplex& right) {
    return hipCmul(left, right);
}

// Умножение CUDA double комплексного числа на вещественное double число (left — комплексное CUDA число, right — вещественное double число)
__device__ hipDoubleComplex operator*(const hipDoubleComplex& left, const double& right) {
    return hipCmul(left, make_hipDoubleComplex(right, 0));
}

// Умножение вещественного double числа на CUDA double комплексное число (left — вещественное double число, right — комплексное CUDA число)
__device__ hipDoubleComplex operator*(const double& left, const hipDoubleComplex& right) {
    return hipCmul(make_hipDoubleComplex(left, 0), right);
}

// Экспонента, принимающая в качестве аргумента CUDA double комплексное число
__device__ hipDoubleComplex exp(const hipDoubleComplex& value) {
    return exp(value.x) * make_hipDoubleComplex(cos(value.y), sin(value.y));
}

// Нахождение оптимального количества нитей в блоке (N — количество точек по одной оси)
int getNumberThreads(int N) {
    auto result = static_cast<int>(round(sqrt(N)));
    while (((N % result) != 0) || (result * result > 1024)) {
        result--;
    }
    return result;
}

// Функция отображения прогресса выполнения преобразования (now — сколько сейчас выполнено операций, max — общее кол-во операций)
__device__ void processing(int now, int max) {
    double percent;
    if (now == max) {
        percent = 100.;
    }
    else {
        percent = trunc(10000. * (static_cast<double>(now) / max)) / 100;
    }
    printf("\rВыполнено %2.2f%", percent);
}

// Обобщенное ядро вычисления интеграла Коллинза (output — массив результата (выходное поле), input — входное поле, x1 — вектор x, x2 — вектор y, x3 — вектор u или z, x4 — вектор v, parameters — массив параметров преобразования, dimension — массив размерностей, progress — атомарная переменная прогресса процесса вычисления интеграла Коллинза, transformType (вид преобразования: 0 — дробное преобразование Фурье, 1 — преобразование Френеля, 2 — другое (с заданной определенной ABCD матрицей)), OxyCrossSection (выбранное сечение: 1 — сечение в плоскости Ouv (поперечное), 0 — сечение в плоскости Ovz (продольное)))
__global__ void collinsKernel(hipDoubleComplex* output, const hipDoubleComplex* input, const double* x1, const double* x2, const double* x3, const double* x4, const double* parameters, const int* dimension, int* progress, int transformType, bool OxyCrossSection)
{
    auto pi = 3.14159265358979323846;
    auto q = blockIdx.x * blockDim.x + threadIdx.x;
    auto p = blockIdx.y * blockDim.y + threadIdx.y;
    auto hx = x1[1] - x1[0];
    auto hy = x2[0] - x2[1];
    auto wavelength = parameters[0];
    auto z = OxyCrossSection ? parameters[1] : x3[q];
    auto u = OxyCrossSection ? x3[q] : parameters[1];
    auto f = !transformType ? parameters[2] : 0;
    auto k = 2 * pi / wavelength;
    auto n1 = dimension[0];
    auto n2 = dimension[1];
    auto n3 = OxyCrossSection ? dimension[1] : dimension[2];
    auto A = 0.0;
    auto B = 0.0;
    auto D = 0.0;
    switch (transformType) {
    case 0:
        A = cos(pi * z / (2 * f));
        B = f * sin(pi * z / (2 * f));
        D = cos(pi * z / (2 * f));
        break;
    case 1:
        A = 1.;
        B = z;
        D = 1.;
        break;
    default:
        A = OxyCrossSection ? parameters[1] : parameters[2];
        B = OxyCrossSection ? parameters[2] : parameters[3];
        D = OxyCrossSection ? parameters[3] : parameters[4];
    }
    auto value = make_hipDoubleComplex(0, 0);
    for (auto i = 0; i < n1; i++) {
        for (auto j = 0; j < n1; j++) {
            auto arg = (k / (2 * B)) * (A * (x2[i] * x2[i] + x1[j] * x1[j]) - 2 * (x2[i] * x4[p] + x1[j] * u) + D * (x4[p] * x4[p] + u * u));
            value += input[i * n1 + j] * exp(make_hipDoubleComplex(0, arg));
        }
    }
    atomicAdd(progress, 1);
    processing(*progress, n2 * n3);
    output[p * n3 + q] = make_hipDoubleComplex(0, -(k / (2 * pi * B))) * value * hx * hy;
}

std::vector<std::vector<std::complex<double>>> calculateCollinsCUDA(const std::vector<std::vector<std::complex<double>>>& input, const std::vector<double>& x1, const std::vector<double>& x2, const std::vector<double>& x3, const std::vector<double>& x4, const std::vector<double>& parameters, const std::vector<int>& dimension, int transformType)
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    bool OxyCrossSection = dimension.size() == 3 ? false : true;

    // Allocate GPU buffers for vectors.

    auto n1 = dimension.at(0);
    auto n2 = dimension.at(1);
    auto n3 = OxyCrossSection ? dimension.at(1) : dimension.at(2);

    auto device_output = new hipDoubleComplex[n2 * n3];
    hipDoubleComplex* device_pointer_output = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_output, static_cast<unsigned long long>(n2) * n3 * sizeof(hipDoubleComplex));
    
    auto device_input = new hipDoubleComplex[input.size() * input.at(0).size()];
    for (auto i = 0; i < input.size(); i++) {
        for (auto j = 0; j < input.at(0).size(); j++) {
            device_input[i * n1 + j] = make_hipDoubleComplex(input.at(i).at(j).real(), input.at(i).at(j).imag());
        }
    }
    hipDoubleComplex* device_pointer_input = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_input, static_cast<unsigned long long>(n1) * n1 * sizeof(hipDoubleComplex));
    cudaStatus = hipMemcpy(device_pointer_input, device_input, static_cast<unsigned long long>(n1) * n1 * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    
    std::vector<double> host_x1 = x1;
    double* pointer_x1 = host_x1.data();
    double* device_pointer_x1 = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_x1, n1 * sizeof(double));
    cudaStatus = hipMemcpy(device_pointer_x1, pointer_x1, n1 * sizeof(double), hipMemcpyHostToDevice);

    std::vector<double> host_x2 = x2;
    double* pointer_x2 = host_x2.data();
    double* device_pointer_x2 = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_x2, n1 * sizeof(double));
    cudaStatus = hipMemcpy(device_pointer_x2, pointer_x2, n1 * sizeof(double), hipMemcpyHostToDevice);

    std::vector<double> host_x3 = x3;
    double* pointer_x3 = host_x3.data();
    double* device_pointer_x3 = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_x3, n3 * sizeof(double));
    cudaStatus = hipMemcpy(device_pointer_x3, pointer_x3, n3 * sizeof(double), hipMemcpyHostToDevice);

    std::vector<double> host_x4 = x4;
    double* pointer_x4 = host_x4.data();
    double* device_pointer_x4 = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_x4, n2 * sizeof(double));
    cudaStatus = hipMemcpy(device_pointer_x4, pointer_x4, n2 * sizeof(double), hipMemcpyHostToDevice);

    std::vector<double> host_parameters = parameters;
    double* pointer_parameters = host_parameters.data();
    double* device_pointer_parameters = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_parameters, host_parameters.size() * sizeof(double));
    cudaStatus = hipMemcpy(device_pointer_parameters, pointer_parameters, host_parameters.size() * sizeof(double), hipMemcpyHostToDevice);

    std::vector<int> host_dimension = dimension;
    int* pointer_dimension = host_dimension.data();
    int* device_pointer_dimension = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_dimension, host_dimension.size() * sizeof(int));
    cudaStatus = hipMemcpy(device_pointer_dimension, pointer_dimension, host_dimension.size() * sizeof(int), hipMemcpyHostToDevice);
    
    int* device_pointer_progress = 0;
    cudaStatus = hipMalloc((void**)&device_pointer_progress, sizeof(int));

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsPerBlock(getNumberThreads(n3), getNumberThreads(n2));
    dim3 numBlocks(n3 / threadsPerBlock.x, n2 / threadsPerBlock.y);
    collinsKernel<<<numBlocks, threadsPerBlock>>>(device_pointer_output, device_pointer_input, device_pointer_x1, device_pointer_x2, device_pointer_x3, device_pointer_x4, device_pointer_parameters, device_pointer_dimension, device_pointer_progress, transformType, OxyCrossSection);
    
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\ncollinsKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        freeGPUMemory(device_pointer_input, device_pointer_output, device_pointer_x1, device_pointer_x2, device_pointer_x3, device_pointer_x4, device_pointer_parameters, device_pointer_dimension, device_pointer_progress);
        throw std::runtime_error("Запуск ядра CUDA не удался!");
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\nhipDeviceSynchronize returned error code %d after launching collinsKernel!\n", cudaStatus);
        freeGPUMemory(device_pointer_input, device_pointer_output, device_pointer_x1, device_pointer_x2, device_pointer_x3, device_pointer_x4, device_pointer_parameters, device_pointer_dimension, device_pointer_progress);
        throw std::runtime_error("Синхронизация данных между хостом и устройством завершилась неудачей!");
    }

    std::cout << "\rВыполнено 100.00%" << std::endl;

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(device_output, device_pointer_output, static_cast<unsigned long long>(n2) * n3 * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        freeGPUMemory(device_pointer_input, device_pointer_output, device_pointer_x1, device_pointer_x2, device_pointer_x3, device_pointer_x4, device_pointer_parameters, device_pointer_dimension, device_pointer_progress);
        throw std::runtime_error("Копирование результата в ОЗУ завершилось неудачей!");
    }

    freeGPUMemory(device_pointer_input, device_pointer_output, device_pointer_x1, device_pointer_x2, device_pointer_x3, device_pointer_x4, device_pointer_parameters, device_pointer_dimension, device_pointer_progress);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("hipDeviceReset failed!");
    }

    std::vector<std::vector<std::complex<double>>> result;
    result.reserve(x4.size());
    for (auto i = 0; i < x4.size(); i++) {
        auto row = std::vector<std::complex<double>>();
        row.reserve(x4.size());
        for (auto j = 0; j < x3.size(); j++) {
            row.emplace_back(std::complex<double>(device_output[i * n3 + j].x, device_output[i * n3 + j].y));
        }
        result.emplace_back(row);
    }

    return result;
}