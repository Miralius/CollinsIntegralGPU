#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <complex>
#include "CollinsIntegralGPU.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_complex.h>

using namespace std;

hipError_t collinsWithCuda(hipDoubleComplex* input, hipDoubleComplex* output, double* x, double* y, double* u, double* v, int n1, int n2, double* fieldParameters);

__global__ void collinsKernel(hipDoubleComplex* output, hipDoubleComplex* input, double* x, double* y, double* u, double* v, int n1, int n2, double hx, double hy, double A, double B, double D, double k)
{
    int q = blockIdx.x * blockDim.x + threadIdx.x;
    int p = blockIdx.y * blockDim.y + threadIdx.y;
    hipDoubleComplex value = make_hipDoubleComplex(0, 0);
    for (int i = 0; i < n1; i++) {
        for (int j = 0; j < n1; j++) {
            double arg = (k / (2 * B)) * (A * (y[i] * y[i] + x[j] * x[j]) - 2 * (y[i] * v[p] + x[j] * u[q]) + D * (v[p] * v[p] + u[q] * u[q]));
            value = hipCadd(value, hipCmul(input[i * n1 + j], make_hipDoubleComplex(cos(arg), sin(arg))));
        }
    }
    output[p * n2 + q] = hipCmul(make_hipDoubleComplex(0, -(k / (2 * 3.14159265358979323846 * B))), hipCmul(value, make_hipDoubleComplex(hx * hy, 0)));
}

vector<vector<complex<double>>> calculateCollinsCUDA(vector<vector<complex<double>>>& inputFunction, vector<double>& x1, vector<double>& x2, vector<double>& x3, vector<double>& x4, int n1, int n2, double waveNumber, vector<double> limits, vector<vector<double>> matrixABCD)
{
    auto input = new hipDoubleComplex[inputFunction.size() * inputFunction.at(0).size()];
    for (auto i = 0; i < inputFunction.size(); i++) {
    	for (auto j = 0; j < inputFunction.at(0).size(); j++) {
    		input[i * n1 + j] = make_hipDoubleComplex(inputFunction.at(i).at(j).real(), inputFunction.at(i).at(j).imag());
    	}
    }

    auto output = new hipDoubleComplex[inputFunction.size() * inputFunction.at(0).size()];

    auto x = new double[x1.size()];
	for (auto i = 0; i < x1.size(); i++) {
		x[i] = x1.at(i);
	}

	auto y = new double[x2.size()];
	for (auto i = 0; i < x2.size(); i++) {
		y[i] = x2.at(i);
	}

	auto u = new double[x3.size()];
	for (auto i = 0; i < x3.size(); i++) {
		u[i] = x3.at(i);
	}

	auto v = new double[x4.size()];
	for (auto i = 0; i < x4.size(); i++) {
		v[i] = x4.at(i);
	}

    auto fieldParameters = new double[6];
    fieldParameters[0] = 2 * limits.at(0) / n1;
    fieldParameters[1] = 2 * limits.at(1) / n1;
    fieldParameters[2] = matrixABCD.at(0).at(0);
    fieldParameters[3] = matrixABCD.at(0).at(1);
    fieldParameters[4] = matrixABCD.at(1).at(1);
    fieldParameters[5] = waveNumber;

    // Add vectors in parallel.
    hipError_t cudaStatus = collinsWithCuda(input, output, x, y, u, v, n1, n2, fieldParameters);
    if (cudaStatus != hipSuccess) {
        error("collinsWithCuda failed!");
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        error("hipDeviceReset failed!");
    }

    vector<vector<complex<double>>> result;
    for (auto i = 0; i < inputFunction.size(); i++) {
        result.push_back(vector<complex<double>>());
        for (auto j = 0; j < inputFunction.at(0).size(); j++) {
            result.back().push_back(complex<double>(output[i * n2 + j].x, output[i * n2 + j].y));
        }
    }

    return result;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t collinsWithCuda(hipDoubleComplex* input, hipDoubleComplex* output, double* x, double* y, double* u, double* v, int n1, int n2, double* fieldParameters)
{
    hipDoubleComplex* dev_in = 0;
    hipDoubleComplex* dev_out = 0;
    size_t pitch_in;
    size_t pitch_out;
    double* dev_x = 0;
    double* dev_y = 0;
    double* dev_u = 0;
    double* dev_v = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output).
    cudaStatus = hipMallocPitch((void**)&dev_in, &pitch_in, n1 * sizeof(hipDoubleComplex), n1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMallocPitch((void**)&dev_out, &pitch_out, n2 * sizeof(hipDoubleComplex), n2);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_x, n1 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_y, n1 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_u, n2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_v, n2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy2D(dev_in, pitch_in, input, n1 * sizeof(hipDoubleComplex), n1 * sizeof(hipDoubleComplex), n1, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_x, x, n1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_y, y, n1 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_u, u, n2 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_v, v, n2 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(n2 / threadsPerBlock.x, n2 / threadsPerBlock.y);
    collinsKernel << <numBlocks, threadsPerBlock >> > (dev_out, dev_in, dev_x, dev_y, dev_u, dev_v, n1, n2, fieldParameters[0], fieldParameters[1], fieldParameters[2], fieldParameters[3], fieldParameters[4], fieldParameters[5]);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_out, n2 * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_in);
    hipFree(dev_out);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_u);
    hipFree(dev_v);

    return cudaStatus;
}